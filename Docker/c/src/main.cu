#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloFromGPU(){
    printf("Hello World from GPU!\n");
}

int main( int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU <<<1, 10>>>();
    hipDeviceReset();
    return 0;
}
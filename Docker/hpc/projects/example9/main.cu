#include "../utils/common.h"

#include <hip/hip_runtime.h>

#include <iostream>
#include <numeric>
#include <functional>
#include <algorithm>

__host__ std::size_t suitableDevice(std::function<uint32_t(const hipDeviceProp_t&)> suitableScore) {
  int max_device_count = 0;
  CHECK(hipGetDeviceCount(&max_device_count));
  if (max_device_count <= 1) return 0;

  std::vector<hipDeviceProp_t> props(max_device_count);
  std::vector<uint32_t> scores(max_device_count);
  uint32_t idx = 0;
  for(auto& prop : props) CHECK(hipGetDeviceProperties(&prop, idx++));

  idx = 0;
  for(auto& score : scores) score = suitableScore(props[idx++]);

  auto it = std::max_element(std::begin(scores), std::end(scores));
  return std::distance(std::begin(scores), it);
}

int main(int argc, char **argv) {
  std::cout << argv[0] << "Starting ... " << std::endl;

  int device_count;
  CHECK(hipGetDeviceCount(&device_count));

  if(device_count == 0)
    std::cout << "There are no available device(s) that support CUDA" << std::endl;
  else
    std::cout << "Detected " << device_count << " CUDA capable device(s)" << std::endl;

  const auto use_device = suitableDevice([](const hipDeviceProp_t& prop) ->uint32_t {
    return prop.multiProcessorCount;
  });
  CHECK(hipSetDevice(use_device));
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, use_device);

  std::cout << "Using Device " << use_device << ":" << prop.name << std::endl;

}
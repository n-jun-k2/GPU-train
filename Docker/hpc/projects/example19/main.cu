#include "../utils/common.h"
#include "../utils/rand_generator_kernel.cuh"
#include "../utils/sync_example_kernel.cuh"

#include <iostream>
#include <cstdlib>
#include <chrono>
#include <string>

std::string sharedMemConfigToString(hipSharedMemConfig config){
  switch (config)
  {
  case hipSharedMemBankSizeDefault:
    return std::string("hipSharedMemBankSizeDefault");
  case hipSharedMemBankSizeEightByte:
    return std::string("hipSharedMemBankSizeEightByte");
  case hipSharedMemBankSizeFourByte:
    return std::string("hipSharedMemBankSizeFourByte");
  }
  return std::string();
}

int main(int argc, char **argv){

  const auto useDevice = suitableDeviceIndex([](const hipDeviceProp_t& prop) ->uint32_t {
    return prop.multiProcessorCount;
  });

  CHECK(hipSetDevice(useDevice));
  hipDeviceProp_t prop;
  CHECK(hipGetDeviceProperties(&prop, useDevice));
  std::cout << prop.name << std::endl;

  /*Access mode*/
  hipSharedMemConfig sharedConfig;
  CHECK(hipDeviceGetSharedMemConfig(&sharedConfig));
  std::cout << "get is " << sharedMemConfigToString(sharedConfig) << std::endl;

  CHECK(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  CHECK(hipDeviceGetSharedMemConfig(&sharedConfig));
  std::cout << "set is " << sharedMemConfigToString(sharedConfig) << std::endl;

  /* capacity setting */
  hipFuncCache_t funcCache = hipFuncCachePreferNone;
  CHECK(hipDeviceSetCacheConfig(funcCache));

  const int blocksize = 32;
  const size_t nElem = 1 << 6;

  dim3 block(blocksize, 1);
  dim3 grid((nElem + block.x - 1) / block.x, 1);

  sync_example_kernel<<<grid, block>>>();
  CHECK(hipDeviceSynchronize());

  CHECK(hipDeviceReset());

}
#include "../utils/common.h"
#include "kernel.cuh"

#include <iostream>
#include <hip/hip_runtime.h>

int main(int argc, char **argv){

  const auto useDevice = suitableDeviceIndex([](const hipDeviceProp_t& prop) ->uint32_t {
    return prop.multiProcessorCount;
  });
  CHECK(hipSetDevice(useDevice));

  constexpr int bdim = 32;
  constexpr int isize = 1 << 6;
  dim3 block(bdim, 1);
  dim3 grid(isize / block.x, 1);

  helloFromGPU<<<grid, block>>>();

  CHECK(hipDeviceSynchronize());
}